#include "hip/hip_runtime.h"
/*
 * File:   test_domain.cpp
 * Author: mbianco
 *
 * Created on February 14, 2014, 4:18 PM
 *
 * Test cuda_storage features
 */

#include <stdlib.h>
#include <stdio.h>
#include <iostream>
#include <common/gpu_clone.h>
#include <storage/hybrid_pointer.h>
#include <storage/cuda_storage.h>
#include <common/layout_map.h>

#ifdef __HIPCC__
template <typename T>
__global__
void add_on_gpu(T * ptr, int d1, int d2, int d3) {
    for (int i = 0; i < d1; ++i) {
        for (int j = 0; j < d2; ++j) {
            for (int k = 0; k < d3; ++k) {
                (*ptr)(i,j,k) = -i-j-k;
            }
        }
    }
}
#endif

bool test_cuda_storage() {

    typedef gridtools::base_storage<gridtools::enumtype::Cuda, double, gridtools::layout_map<0,1,2> > storage_type;

    int d1 = 3;
    int d2 = 3;
    int d3 = 3;

    storage_type data(d1,d2,d3,-1, std::string("data"));

    for (int i = 0; i < d1; ++i) {
        for (int j = 0; j < d2; ++j) {
            for (int k = 0; k < d3; ++k) {
                data(i,j,k) = i+j+k;
#ifndef NDEBUG
                std::cout << data(i,j,k) << " ";
#endif
            }
#ifndef NDEBUG
            std::cout << std::endl;
#endif
        }
#ifndef NDEBUG
        std::cout << std::endl;
        std::cout << std::endl;
#endif
    }

    data.h2d_update();
    data.clone_to_gpu();

#ifdef __HIPCC__
    add_on_gpu<<<1,1>>>(data.gpu_object_ptr, d1, d2, d3);
    hipDeviceSynchronize();
#endif
    data.d2h_update();

    bool same = true;
    for (int i = 0; i < d1; ++i) {
        for (int j = 0; j < d2; ++j) {
            for (int k = 0; k < d3; ++k) {
#ifndef NDEBUG
                std::cout << data(i,j,k) << " ";
#endif
                if (data(i,j,k) != -i-j-k)
                    same = false;
            }
#ifndef NDEBUG
            std::cout << std::endl;
#endif
        }
#ifndef NDEBUG
        std::cout << std::endl;
        std::cout << std::endl;
#endif
    }

    return same;
}
