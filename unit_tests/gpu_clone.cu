#include "hip/hip_runtime.h"
/** This code tests a solution to have clone objects on GPUs. The objects can have references to
    data members that must be initialized on GPU with references on the device

    Authors: Mauro Bianco, Ugo Varetto

    This version uses a gpu enabled boost::fusion library
*/


#define BOOST_NO_CXX11_RVALUE_REFERENCES

#include <boost/fusion/include/vector.hpp>
#include <boost/fusion/include/zip_view.hpp>
#include <boost/fusion/include/for_each.hpp>

#include <stdio.h>
#include <stdlib.h>

#include <gpu_clone.h>

/********************************************************
    GENERIC CODE THAW WORKS WITH ANY (almost POD) OBJECT
*********************************************************/


/********************************************************
    SPECIFIC CODE WITH AN OBJECT THAT HAS REFERENCES
    BUT NEED TO BE CLONED ON GPU
*********************************************************/

struct A: public gridtools::gpu_clone<A> {
    typedef boost::fusion::vector<int, double> v_type;
    v_type v1;
    v_type v2;

    typedef boost::fusion::vector<v_type&, v_type&> support_t;
    typedef boost::fusion::zip_view<support_t> zip_view_t;

    zip_view_t zip_view;

    A(v_type& a, v_type& b)
        : v1(a)
        , v2(b)
        , zip_view(support_t(v1, v2))
    {
        clone();
    }

    __device__
    A(A const& a) 
        : v1(a.v1)
        , v2(a.v2)
        , zip_view(support_t(v1, v2))
    { }

    ~A() { }

    void update_gpu_copy() {
        clone();
    }

    __host__ __device__
    void out() const {
        printf("v1:  ");
        boost::fusion::for_each(v1, print_elements());
        printf("\n");

        printf("v2:  ");
        boost::fusion::for_each(v2, print_elements());
        printf("\n");

        printf("zip: ");
        boost::fusion::for_each(zip_view, print_zip());
        printf("\n");
    }

private:
    struct print_elements {
        __host__ __device__
        void operator()(int u) const {
            printf("%d, ", u);
        }

        __host__ __device__
        void operator()(double u) const {
            printf("%e, ", u);
        }
    };

    struct print_zip {
        template <typename V>
        __host__ __device__
        void operator()(V const & v) const {
            boost::fusion::for_each(v, print_elements());
            printf("\n");
        }
    };

};

__global__
void print_on_gpu(A * a) {
    a->out();
}


struct minus1 {
    template <typename T>
    __host__ __device__ // Avoid warning
    void operator()(T & x) const {
        x -= 1;
    }
};

int main(int argc, char** argv) {

    if (argc != 2) {
        printf("Multiplicator is needed\n");
        return 1;
    }

    int m = atoi(argv[1]);

    typename A::v_type w1(m*1, m*3.1415926);
    typename A::v_type w2(m*2, m*2.7182818);

    A a(w1, w2);

    a.out();

    printf("Performing the same operation on GPU on cloned object\n");

    print_on_gpu<<<1,1>>>(a.gpu_object_ptr);

    hipDeviceSynchronize();

    printf("Updating the object with -1\n");

    boost::fusion::for_each(a.v1, minus1());
    boost::fusion::for_each(a.v2, minus1());
    a.update_gpu_copy();

    a.out();

    printf("Performing the same operation on GPU on cloned object\n");

    print_on_gpu<<<1,1>>>(a.gpu_object_ptr);

    hipDeviceSynchronize();

    return 0;
}
