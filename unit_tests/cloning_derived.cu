#include <errno.h>
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <iomanip>
#include "common/gpu_clone.hpp"
#include "storage/hybrid_pointer.hpp"

using namespace gridtools;

template <typename t_derived>
struct base : public clonable_to_gpu<t_derived> {
    uint_t m_size;

    base(uint_t s) : m_size(s) {}

    __host__ __device__
    base(base const& other) // default construct clonable_to_gpu
        : m_size(other.m_size)
    {  }
};

template <typename value_type>
struct derived: public base<derived<value_type> > {
    hybrid_pointer<value_type> data;

    derived(uint_t s)
        : base<derived<value_type> >(s)
        , data(s)
    {
        for (uint_t i = 0; i < data.get_size(); ++i)
            data[i] = data.get_size()-i;
        data.update_gpu();
    }

    __host__ __device__
    derived(derived const& other)
        : base<derived<value_type> >(other)
        , data(other.data)
    {  }

};

int main(int argc, char** argv) {

    if(argc < 2) {
        printf("ERROR: must pass a buffer size.\n\tUsage: %s [buffer size]\n", argv[0]);
        return EXIT_FAILURE;
    }

    char *pend = 0;
    uint_t buffer_size = strtol(argv[1], &pend, 10);
    if(buffer_size == 0 || pend == 0 || *pend != '\0' || errno == ERANGE) {
        printf("ERROR: invalid buffer size.\n\tUsage: %s [buffer size]\n", argv[0]);
        return EXIT_FAILURE;
    }

    int_t res = EXIT_SUCCESS;

    derived<uint_t> a(buffer_size);
    for(uint_t i = 0; i < a.data.get_size(); ++i) {
        if(a.data[i] != buffer_size - i)
            res = EXIT_FAILURE;
    }

    a.clone_to_gpu();
    a.data.update_gpu();

    for(uint_t i = 0; i < a.data.get_size(); ++i) {
        if(a.data[i] != buffer_size - i)
            res = EXIT_FAILURE;
    }

    for(uint_t i = 0; i < a.data.get_size(); ++i) {
        if(a.data[i] != buffer_size - i)
            res = EXIT_FAILURE;
    }

    hipDeviceSynchronize();
    a.clone_from_gpu();
    a.data.update_cpu();

    for(uint_t i = 0; i < a.data.get_size(); ++i) {
        if(a.data[i] != buffer_size - i + 1)
            res = EXIT_FAILURE;
    }

    return res;
}
