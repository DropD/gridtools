#include "hip/hip_runtime.h"
#include <gpu_clone.h>
#include <stdio.h>
#include <string.h>
#include <hybrid_pointer.h>

#ifndef __HIPCC__
#define __host__
#define __device__
#endif

struct B: public gridtools::clonable_to_gpu<B> {
    gridtools::hybrid_pointer<char> pointer_to_use;
    int size;
    int &ref;

    B(int size) : size(size), ref(size), pointer_to_use(size) { }

    __host__ __device__
    B(B const& other)
        : pointer_to_use(other.pointer_to_use)
        , size(other.size)
        , ref(size)
    {}

    ~B() {
        pointer_to_use.free_it();
    }

    void update_gpu() {
        pointer_to_use.update_gpu();
    }

    void update_cpu() {
        pointer_to_use.update_cpu();
    }

};


struct A :public gridtools::clonable_to_gpu<A> {
    int a;
    int &b;
    B p;

    A(int a, int size) : a(a), b(a), p(size) {}

    __host__ __device__
    A(A const& other) 
        : a(other.a)
        , b(a)
        , p(other.p)
    {}
};

#ifdef __HIPCC__
__global__
void test(A* a) {
    printf(">%s<\n", (char*)(a->p.pointer_to_use));
    printf("the reference in A %d\n", a->b);

    a->b++;
    a->p.ref *= 2;
    a->p.pointer_to_use[4] = 'W';
    printf(">%s<\n", &(a->p.pointer_to_use[0]));
}
#endif

void try_char(char *p) {}

int main() {
    char s[30] = "The world will end ... now";

    A a(34, 30);

    // Copy the string to GPU
    memcpy(a.p.pointer_to_use.pointer_to_use, s, 30);
    a.p.update_gpu();

    // Clone a (and b_object) to gpu
    a.clone_to_gpu();

#ifdef __HIPCC__
    test<<<1,1>>>(a.gpu_object_ptr);
#endif
    a.clone_from_gpu();

    printf("%d\n", a.p.size);

    a.p.update_cpu();

    printf("%s\n", (char*)(a.p.pointer_to_use));
    printf("the reference in A %d\n", a.b);

#ifdef __HIPCC__
    hipDeviceSynchronize();
#endif

    return 0;
}
